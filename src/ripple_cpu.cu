#include "hip/hip_runtime.h"
// Significant portions shamelessly stolen from Kandrot & Sanders CUDA by example

#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep
#include "common.h"
#include "nv/cpu_anim.h"

#ifdef _WIN32
#include <chrono>
#include <thread>
#endif
/* assuring that any block size will be divisible by warps size */
#define THREADS_IN_WARP 32
#define TILE_WIDTH ((THREADS_IN_WARP) - 2)

#define SCREEN_WIDTH 1024
#define SCREEN_HEIGHT 768

#define TILE_DIM 32

struct DataBlock {
    unsigned char *dev_bitmap;
    CPUAnimBitmap *bitmap;
};

void cleanup(DataBlock *d) {
    hipFree(d->dev_bitmap);
}

__global__ void compute_ripple_bitmap(unsigned char * bitmap, int ticks)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset = x + y * blockDim.x * gridDim.x;

    float fx = x - SCREEN_WIDTH/2;
    float fy = y - SCREEN_HEIGHT/2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char grey = (unsigned char) (128.0f + 127.0f * cos(d/10.0f - ticks/7.0f) / 
                                                            (d/10.0f + 1.0f));

    bitmap[offset*4 + 0] = grey;
    bitmap[offset*4 + 1] = grey;
    bitmap[offset*4 + 2] = grey;
    bitmap[offset*4 + 3] = 255;
}

void generate_frame(DataBlock * d, int ticks) {
    dim3 grids(SCREEN_WIDTH/TILE_DIM, SCREEN_HEIGHT/TILE_DIM);
    dim3 threads(TILE_DIM, TILE_DIM);
    compute_ripple_bitmap<<<grids, threads>>>(d->dev_bitmap, ticks);

    gpuErrchk(hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost));
}

int main(int argc, char **argv) {    
    DataBlock data;
    CPUAnimBitmap bitmap(SCREEN_WIDTH, SCREEN_HEIGHT, &data);
    data.bitmap = &bitmap;

    gpuErrchk(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
    

    bitmap.anim_and_exit((void (*)(void*,int))generate_frame, (void(*)(void*))cleanup);
}
