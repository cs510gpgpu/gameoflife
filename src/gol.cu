#include "hip/hip_runtime.h"
/* Original Author: Christopher Mitchell <chrism@lclark.edu>
 * Date: 2011-07-15
 * Heavy modification by Mitch Souders <msouders@cs.pdx.edu> as part of a homework assignment
 * to compute the Game Of Life using CUDA.
 */

#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep
#include "common.h"
#include "nv/gpu_anim.h"

#ifdef _WIN32
#include <chrono>
#include <thread>
#endif
/* assuring that any block size will be divisible by warps size */
#define THREADS_IN_WARP 32
#define TILE_WIDTH ((THREADS_IN_WARP) - 2)

const int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
                           {-1, 0},       {1, 0},
                           {-1,-1},{0,-1},{1,-1}};

__constant__ int cuda_offsets[8][2];

void fill_board(int *board, int elements) {
    int i;
    for (i=0; i<elements; i++)
        board[i] = (rand() % 3) >> 1;
		//This fix is nessary since windows rand() produces a number whose least siginificant bit
		//	repeats every 161072 causing patterns to emerge and end the simulation more quickly
		//  on some screen resolutions. 
}

void print_board(int *board, int WIDTH, int HEIGHT) {
    int x, y;
    for (y=0; y<HEIGHT; y++) {
        for (x=0; x<WIDTH; x++) {
            char c = board[y * WIDTH + x] ? '#':' ';
            printf("%c", c);
        }
        printf("\n");
    }
    printf("-----\n");
}

/* simply swaps pointers for two boards when they need to be
   in different locations */
void swap_board(int ** b1, int ** b2)
{
    int * tmp = *b1;
    *b1 = *b2;
    *b2 = tmp;
}

void step(int * next, int * current, int WIDTH, int HEIGHT) {
    // coordinates of the cell we're currently evaluating
    int x, y;
    // offset index, neighbor coordinates, alive neighbor count
    int i, nx, ny, num_neighbors;

    // write the next board state
    for (y=0; y<HEIGHT; y++) {
        for (x=0; x<WIDTH; x++) {
            // count this cell's alive neighbors
            num_neighbors = 0;
            for (i=0; i<8; i++) {
                // To make the board torroidal, we use modular arithmetic to
                // wrap neighbor coordinates around to the other side of the
                // board if they fall off.
                nx = (x + offsets[i][0] + WIDTH) % WIDTH;
                ny = (y + offsets[i][1] + HEIGHT) % HEIGHT;
                if (current[ny * WIDTH + nx]) {
                    num_neighbors++;
                }
            }

            // apply the Game of Life rules to this cell
            next[y * WIDTH + x] = 0;
            if ((current[y * WIDTH + x] && num_neighbors==2) ||
                    num_neighbors==3) {
                next[y * WIDTH + x] = 1;
            }
        }
    }
}

/* the naive approach is more or less a copy/pasted version of the original algorithm
   which has minor changes to work within a kernel space */
__global__ void naive_compute_gol(int * next, int * board, int WIDTH, int HEIGHT)
{    
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= WIDTH || y >= HEIGHT) {
        return;
    }
    
    int num_neighbors = 0;
    for (int i=0; i<8; i++) {
        int nx = (x + cuda_offsets[i][0] + WIDTH) % WIDTH;
        int ny = (y + cuda_offsets[i][1] + HEIGHT) % HEIGHT;
        if (board[ny * WIDTH + nx]) {
            num_neighbors++;
        }
    }
    
    next[y * WIDTH + x] = 0;
    if ((board[y * WIDTH + x] && num_neighbors==2) ||
        num_neighbors==3) {
        next[y * WIDTH + x] = 1;
    }
}

/* 
The tiled approach maintains some shared memory to keep the neighboards locally stored
for the given block. This reduces the memory overhead.
Additionally this approach uses a manually unrolled loop iteration to remove dependency
on constant memory for the offsets.
 */
__global__ void tile_compute_gol(int * next, int * board, int WIDTH, int HEIGHT)
{
    __shared__ int tile[TILE_WIDTH + 2][TILE_WIDTH + 2];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tileX = (blockIdx.x * (blockDim.x - 2)) + (threadIdx.x -1);
    int tileY = (blockIdx.y * (blockDim.y - 2)) + (threadIdx.y -1);

    /* fill the tile board with each item */
    tile[threadIdx.x][threadIdx.y] =
        board[((tileY + HEIGHT) % HEIGHT) * WIDTH + ((tileX + WIDTH) % WIDTH)];

    if (threadIdx.x == 0 || threadIdx.y == 0 || threadIdx.x > TILE_WIDTH  || threadIdx.y > TILE_WIDTH) {
        return; /* these threads do not contribute to answer */
    } else if (tileX >= WIDTH || tileY >= HEIGHT) {
        return; /* these are threads that extend over the edge since the board is not evenly divided by blocks */
    }

    __syncthreads(); /* all spots in `tile' should be full after sync */
    int num_neighbors = 0;
    /* does this manual loop unrolling improve speed over accessing the
       constants in the constant memory? worth a shot */
    num_neighbors += tile[tx +-1][ty + 1];
    num_neighbors += tile[tx + 0][ty + 1];
    num_neighbors += tile[tx + 1][ty + 1];
    num_neighbors += tile[tx +-1][ty + 0];
    num_neighbors += tile[tx + 1][ty + 0];
    num_neighbors += tile[tx +-1][ty +-1];
    num_neighbors += tile[tx + 0][ty +-1];
    num_neighbors += tile[tx + 1][ty +-1];

    /* compute final result */
    next[tileY * WIDTH + tileX] = (
        (tile[threadIdx.x][threadIdx.y] && num_neighbors==2)
        || num_neighbors==3);
}


__global__ void tile_compute_gol_bitmap(uchar4* bitmap, int * next, int * board, int WIDTH, int HEIGHT)
{
    __shared__ int tile[TILE_WIDTH + 2][TILE_WIDTH + 2];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tileX = (blockIdx.x * (blockDim.x - 2)) + (threadIdx.x -1);
    int tileY = (blockIdx.y * (blockDim.y - 2)) + (threadIdx.y -1);

    /* fill the tile board with each item */
    tile[threadIdx.x][threadIdx.y] =
        board[((tileY + HEIGHT) % HEIGHT) * WIDTH + ((tileX + WIDTH) % WIDTH)];

    if (threadIdx.x == 0 || threadIdx.y == 0 || threadIdx.x > TILE_WIDTH  || threadIdx.y > TILE_WIDTH) {
        return; /* these threads do not contribute to answer */
    } else if (tileX >= WIDTH || tileY >= HEIGHT) {
        return; /* these are threads that extend over the edge since the board is not evenly divided by blocks */
    }

    __syncthreads(); /* all spots in `tile' should be full after sync */
    int num_neighbors = 0;
    /* does this manual loop unrolling improve speed over accessing the
       constants in the constant memory? worth a shot */
    num_neighbors += tile[tx +-1][ty + 1];
    num_neighbors += tile[tx + 0][ty + 1];
    num_neighbors += tile[tx + 1][ty + 1];
    num_neighbors += tile[tx +-1][ty + 0];
    num_neighbors += tile[tx + 1][ty + 0];
    num_neighbors += tile[tx +-1][ty +-1];
    num_neighbors += tile[tx + 0][ty +-1];
    num_neighbors += tile[tx + 1][ty +-1];

    int offset = tileY * WIDTH + tileX;
    /* compute final result */
    next[offset] = (
        (tile[threadIdx.x][threadIdx.y] && num_neighbors==2)
        || num_neighbors==3);

    bitmap[offset].x = next[offset] ? 255 : 0;
    bitmap[offset].y = next[offset] ? 255 : 0;
    bitmap[offset].z = next[offset] ? 255 : 0;
    bitmap[offset].w = 255;
}
    
/* returns true if boards are equivalent */
bool compare_board(int * b1, int * b2, int len)
{
    for (int i = 0; i < len; i++) {
        if (b1[i] != b2[i]) {
            return false;
        }
    }
    return true;
}

/* copies a board from src to target */
void copy_board(int * target, int * src, int len)
{
    memcpy(target, src, len * sizeof(int));
}

/* the naive game of life, which uses constant memory for offsets */
void gol_naive_device(int * board, int iterations, int WIDTH, int HEIGHT)
{
    int * dev_board;
    int * dev_next;
    int i;
    gpuErrchk(hipMalloc((void **) &dev_board, sizeof(int) * WIDTH * HEIGHT));
    gpuErrchk(hipMalloc((void **) &dev_next, sizeof(int) * WIDTH * HEIGHT));
    
    gpuErrchk(hipMemcpy(dev_board, board, sizeof(int) * WIDTH * HEIGHT, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cuda_offsets), offsets, sizeof(cuda_offsets) ));

    dim3 threadsPerBlock(THREADS_IN_WARP, THREADS_IN_WARP);
    dim3 numBlocks(ceil((float)WIDTH / THREADS_IN_WARP),ceil((float)HEIGHT / THREADS_IN_WARP));

    gpuErrchk( hipDeviceSynchronize() ); /* wait for mem to be copied? */
    for (i = 0; i < iterations; i++) {
        tile_compute_gol<<<numBlocks, threadsPerBlock>>>(dev_next, dev_board, WIDTH, HEIGHT);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() ); /* wait for computation to complete */

        /* swap the two boards to allow memory to already be in the correct location */
        swap_board(&dev_next, &dev_board);
    }

    /* only copy memory from device when the user program wants it (after the iterations) */
    gpuErrchk(hipMemcpy(board, dev_board, sizeof(int) * WIDTH * HEIGHT, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_board));
    gpuErrchk(hipFree(dev_next));
}

/* the tiled game of life management function */
void gol_device(int * board, int iterations, int WIDTH, int HEIGHT)
{
    int * dev_board;
    int * dev_next;
    int i;
    gpuErrchk(hipMalloc((void **) &dev_board, sizeof(int) * WIDTH * HEIGHT));
    gpuErrchk(hipMalloc((void **) &dev_next, sizeof(int) * WIDTH * HEIGHT));
    
    gpuErrchk(hipMemcpy(dev_board, board, sizeof(int) * WIDTH * HEIGHT, hipMemcpyHostToDevice));
    /* this will "over" allocate threads, to fully populate shared memory on each block */
    dim3 threadsPerBlock(TILE_WIDTH + 2, TILE_WIDTH + 2);
    dim3 numBlocks(ceil((float)WIDTH / (TILE_WIDTH)),ceil((float)HEIGHT / (TILE_WIDTH)));

    gpuErrchk( hipDeviceSynchronize() ); /* wait for mem to be copied? */
    for (i = 0; i < iterations; i++) {
        tile_compute_gol<<<numBlocks, threadsPerBlock>>>(dev_next, dev_board, WIDTH, HEIGHT);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() ); /* wait for computation to complete */

        /* swap the two boards to allow memory to already be in the correct location */
        swap_board(&dev_next, &dev_board);
    }

    /* only copy memory from device when the user program wants it (after the iterations) */
    gpuErrchk(hipMemcpy(board, dev_board, sizeof(int) * WIDTH * HEIGHT, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_board));
    gpuErrchk(hipFree(dev_next));
}

void animate(int * board, int WIDTH, int HEIGHT) {
	#ifdef _WIN32
	
	#else
	struct timespec delay = {0, 125000000}; // 0.125 seconds
    struct timespec remaining;
	#endif
	
	int iteration = 0;
    while (1) {
        printf("Iteration: %d\n", iteration++);
        print_board(board, WIDTH, HEIGHT);
        gol_device(board, 1, WIDTH, HEIGHT);
        // We sleep only because textual output is slow and the console needs
        // time to catch up. We don't sleep in the graphical X11 version.
		#ifdef _WIN32
		std::this_thread::sleep_for(std::chrono::milliseconds(125));
		#else
		nanosleep(&delay, &remaining);
		#endif
	}
}

struct DataBlock {
    int HEIGHT;
    int WIDTH;
    int * dev_board;
    int * dev_next;
    hipEvent_t start, stop;
    int frames;
    float totalTime;
};


void anim_gpu( uchar4* outputBitmap, DataBlock *d, int ticks ) {
    gpuErrchk( hipEventRecord( d->start, 0 ) );
    dim3 threadsPerBlock(TILE_WIDTH + 2, TILE_WIDTH + 2);
    dim3 numBlocks(ceil((float)d->WIDTH / (TILE_WIDTH)),ceil((float)d->HEIGHT / (TILE_WIDTH)));
    
    tile_compute_gol_bitmap<<<numBlocks, threadsPerBlock>>>(outputBitmap, d->dev_next, d->dev_board, d->WIDTH, d->HEIGHT);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() ); /* wait for computation to complete */
    gpuErrchk( hipEventRecord( d->stop, 0 ) );
    gpuErrchk( hipEventSynchronize( d->stop ) );
    float   elapsedTime;
    gpuErrchk( hipEventElapsedTime( &elapsedTime, d->start, d->stop ) );
    /* swap the two boards to allow memory to already be in the correct location */
    swap_board(&d->dev_next, &d->dev_board);

    d->totalTime += elapsedTime;
    d->frames++;
    if ((d->frames & 0x7F) == 0) {
        printf( "(%d) Average Time per frame:  %3.1f ms\n",
            d->frames, d->totalTime/d->frames );
    }
}

void anim_exit( DataBlock *d ) {
    gpuErrchk(hipFree(d->dev_board));
    gpuErrchk(hipFree(d->dev_next));
    gpuErrchk( hipEventDestroy( d->start ) );
    gpuErrchk( hipEventDestroy( d->stop ) );
}

void gpu_gameoflife(int WIDTH, int HEIGHT, int * board)
{
    DataBlock   d;
    GPUAnimBitmap bitmap( WIDTH, HEIGHT, &d );

    d.HEIGHT = HEIGHT;
    d.WIDTH = WIDTH;
    d.frames = 0;
    d.totalTime = 0;
    
    gpuErrchk( hipEventCreate( &d.start ) );
    gpuErrchk( hipEventCreate( &d.stop ) );
    
    gpuErrchk(hipMalloc((void **) &d.dev_board, sizeof(int) * WIDTH * HEIGHT));
    gpuErrchk(hipMalloc((void **) &d.dev_next, sizeof(int) * WIDTH * HEIGHT));
    
    gpuErrchk(hipMemcpy(d.dev_board, board, sizeof(int) * WIDTH * HEIGHT, hipMemcpyHostToDevice));

    gpuErrchk( hipDeviceSynchronize() ); /* wait for mem to be copied */
    
    bitmap.anim_and_exit( (void (*)(uchar4*,void*,int))anim_gpu,
        (void (*)(void*))anim_exit );
}

int main(int argc, char *argv[]) {
    int WIDTH = 1024;
    int HEIGHT = 768;
    
	if (argc > 1 && argc <= 3) {
		WIDTH = atoi(argv[1]);
		HEIGHT = atoi(argv[2]);
	}
	
	int elements = WIDTH * HEIGHT;
    
    int * default_board = (int *)malloc(sizeof(int) * elements);
    int * default_next = (int *)malloc(sizeof(int) * elements);

    int * cuda_board = (int *)malloc(sizeof(int) * elements);
    srand(time(NULL));
 
    fill_board(default_board, elements);
    copy_board(cuda_board, default_board, elements);

    gpu_gameoflife(WIDTH, HEIGHT, cuda_board);
    
    // Sanity Check CUDA for 10 Steps (each checked)
    for (int i = 0; i < 10; i++) {
        step(default_next, default_board, WIDTH, HEIGHT);
        gol_device(cuda_board, 1, WIDTH, HEIGHT);
        
        if (!compare_board(cuda_board, default_next, elements)) {
            print_board(cuda_board, WIDTH, HEIGHT);
            print_board(default_next, WIDTH, HEIGHT);
            printf("SANITY: Boards do not match (iteration %d)!\n", i);
            exit(1);
        }

        swap_board(&default_next, &default_board);
    }


    //Sanity Check CUDA with 10 unmonitored (completely on device) steps
    int unmonitored = 10;
    for (int i = 0; i < unmonitored; i++) {
        step(default_next, default_board, WIDTH, HEIGHT);
        swap_board(&default_next, &default_board);
    }
    gol_device(cuda_board, unmonitored, WIDTH, HEIGHT);
    if (!compare_board(cuda_board, default_board, elements)) {
        print_board(cuda_board, WIDTH, HEIGHT);
        print_board(default_board, WIDTH, HEIGHT);
        printf("SANITY: Boards do not match (after 10 on device iterations)!\n");
        exit(1);
    }

    /* cleanup no longer needed boards */
    free(default_board);
    free(default_next);
    
    /* it appears to be sane, run the animation routine */
    animate(cuda_board, WIDTH, HEIGHT);

    free(cuda_board);

    return 0;
}
