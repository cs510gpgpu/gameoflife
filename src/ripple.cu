#include "hip/hip_runtime.h"
// Significant portions shamelessly stolen from Kandrot & Sanders CUDA by example

#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep
#include "common.h"
#include "nv/gpu_anim.h"
#include "nv/cpu_anim.h"

#ifdef _WIN32
#include <chrono>
#include <thread>
#endif

struct GPUDataBlock {
    int HEIGHT;
    int WIDTH;
    int block_width;
};

struct Args_t globalArgs;

__global__ void compute_ripple_bitmap(uchar4* bitmap, int ticks, int WIDTH, int HEIGHT)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (x >= WIDTH || y >= HEIGHT) {
        return;
    }

    int offset = x + y * blockDim.x * gridDim.x;

    float fx = x - WIDTH/2;
    float fy = y - HEIGHT/2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char grey = (unsigned char) (128.0f + 127.0f * cos(d/10.0f - ticks/7.0f) / 
                                                            (d/10.0f + 1.0f));

    bitmap[offset].x = grey;
    bitmap[offset].y = grey;
    bitmap[offset].z = grey;
    bitmap[offset].w = 255;
}

void generate_frame(uchar4 * bitmap, GPUDataBlock * d, int ticks) {
    static int count = 0;
    dim3 grids(ceil((float)d->WIDTH/d->block_width), ceil((float)d->HEIGHT/d->block_width));
    dim3 threads(d->block_width, d->block_width);
    count++;
    compute_ripple_bitmap<<<grids, threads>>>(bitmap, ticks, d->WIDTH, d->HEIGHT);
    timeout(&globalArgs, count);
}

struct CPUDataBlock {
    uchar4 *dev_bitmap;
    CPUAnimBitmap *bitmap;
    int HEIGHT;
    int WIDTH;
    int block_width;
};

void generate_frame_cpu(CPUDataBlock * d, int ticks) {
    dim3 grids(ceil((float)d->WIDTH/d->block_width), ceil((float)d->HEIGHT/d->block_width));
    dim3 threads(d->block_width, d->block_width);
    static int count = 0;
    count++;
    compute_ripple_bitmap<<<grids, threads>>>(d->dev_bitmap, ticks, d->WIDTH, d->HEIGHT);

    gpuErrchk(hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost));
    timeout(&globalArgs, count);
}

void cleanup_cpu(CPUDataBlock *d) {
    hipFree(d->dev_bitmap);
}

int main(int argc, char **argv) {    
    processArgs("ripple", argv, argc, &globalArgs);

    int WIDTH = globalArgs.width;
    int HEIGHT = globalArgs.height;
    int block_width = globalArgs.blockwidth;
    MODES mode = globalArgs.mode;

    switch(mode) {
    case PROFILE_NONE:
        printf("Set a profile mode. \"None\" is unimplemented.\n");
        break;
    case PROFILE_GPU:
        {
            GPUDataBlock data;            
            data.HEIGHT = HEIGHT;
            data.WIDTH = WIDTH;
            data.block_width = block_width;
        	GPUAnimBitmap bitmap(data.WIDTH, data.HEIGHT, &data);
            bitmap.anim_and_exit((void (*)(uchar4*,void*,int))generate_frame, NULL);
        }
        break;
    case PROFILE_CPU:
        {
            CPUDataBlock data;
            data.HEIGHT = HEIGHT;
            data.WIDTH = WIDTH;
            data.block_width = block_width;
            CPUAnimBitmap bitmap(data.WIDTH, data.HEIGHT, &data);
            data.bitmap = &bitmap;
            gpuErrchk(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));
            bitmap.anim_and_exit((void (*)(void*,int))generate_frame_cpu, (void(*)(void*))cleanup_cpu);
        }
        break;
    default:
        printf("Unhandled mode by ripple.\n");
        exit(1);
    }
}
