#include "hip/hip_runtime.h"
// Significant portions shamelessly stolen from Kandrot & Sanders CUDA by example

#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep
#include "common.h"
#include "nv/gpu_anim.h"
#include "nv/cpu_anim.h"

#ifdef _WIN32
#include <chrono>
#include <thread>
#endif

struct GPUDataBlock {
    int HEIGHT;
    int WIDTH;
    int block_width;
};

struct Args_t globalArgs;

__global__ void compute_ripple_bitmap(uchar4* bitmap, int ticks, int WIDTH, int HEIGHT)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (x >= WIDTH || y >= HEIGHT) {
        return;
    }

    int offset = x + y * blockDim.x * gridDim.x;

    float fx = x - WIDTH/2;
    float fy = y - HEIGHT/2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char grey = (unsigned char) (128.0f + 127.0f * cos(d/10.0f - ticks/7.0f) / 
                                                            (d/10.0f + 1.0f));

    bitmap[offset].x = grey;
    bitmap[offset].y = grey;
    bitmap[offset].z = grey;
    bitmap[offset].w = 255;
}

void generate_frame(uchar4 * bitmap, GPUDataBlock * d, int ticks) {
    static int count = 0;
    dim3 grids(ceil((float)d->WIDTH/d->block_width), ceil((float)d->HEIGHT/d->block_width));
    dim3 threads(d->block_width, d->block_width);
    count++;
    compute_ripple_bitmap<<<grids, threads>>>(bitmap, ticks, d->WIDTH, d->HEIGHT);
    timeout(&globalArgs, count);
}

struct CPUDataBlock {
    GPUDataBlock gpu;
    uchar4 *dev_bitmap;
    CPUAnimBitmap *bitmap;
};

void generate_frame_cpu(CPUDataBlock * d, int ticks) {
    generate_frame(d->dev_bitmap, &d->gpu, ticks);
    gpuErrchk(hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost));
}

void cleanup_gpu(GPUDataBlock *d)
{
}

void cleanup_cpu(CPUDataBlock *d) {
    cleanup_gpu(&d->gpu);
    hipFree(d->dev_bitmap);
}

void init_gpu(GPUDataBlock *d)
{
    d->HEIGHT = globalArgs.height;
    d->WIDTH = globalArgs.width;
    d->block_width = globalArgs.blockwidth;
}

int main(int argc, char **argv) {    
    processArgs("ripple", argv, argc, &globalArgs);
    switch(globalArgs.mode) {
    case PROFILE_NONE:
        printf("Set a profile mode. \"None\" is unimplemented.\n");
        break;
    case PROFILE_GPU:
        {
            GPUDataBlock data;            
            init_gpu(&data);
        	GPUAnimBitmap bitmap(data.WIDTH, data.HEIGHT, &data);
            bitmap.anim_and_exit((void (*)(uchar4*,void*,int))generate_frame, NULL);
        }
        break;
    case PROFILE_CPU:
        {
            CPUDataBlock data;
            init_gpu(&data.gpu);
            CPUAnimBitmap bitmap(data.gpu.WIDTH, data.gpu.HEIGHT, &data);
            data.bitmap = &bitmap;
            gpuErrchk(hipMalloc((void**)&data.dev_bitmap, data.bitmap->image_size()));
            bitmap.anim_and_exit((void (*)(void*,int))generate_frame_cpu, (void(*)(void*))cleanup_cpu);
        }
        break;
    default:
        printf("Unhandled mode by ripple.\n");
        exit(1);
    }
}
