#include "common.h"
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include "argtable3.h"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void timeout(Args args, int frames) {
    printf("%d frames\n", frames);
    if (args->timeout != 0 && args->timeout < frames) {
        hipDeviceReset();
        exit(0);
    }
}

int processArgs(const char * progname, char ** argv, int argc, Args args)
{
    struct arg_lit *help, *profile;
    struct arg_int *height, *width, *block_width, *timeout;
    struct arg_str *mode;
    struct arg_end *end;

    int do_exit = 0;
    void *argtable[] = {
        help    = arg_litn(NULL, "help",  0, 1, "display this help and exit"),       
        height  = arg_intn(NULL, "height","<n>", 0, 1,   "height"),
        width   = arg_intn(NULL, "width", "<n>",  0, 1, "width"),
        timeout = arg_intn(NULL, "timeout", "<n>",  0, 1, "timeout"),
        block_width = arg_intn(NULL, "blockwidth", "<n>",  0, 1, "block width"),
        mode    = arg_strn(NULL, "mode", "modename",  0, 1, "allowed modes: gpu cpu"),
        profile = arg_litn(NULL, "profile", 0, 1, "disables text output"),
        end     = arg_end(20),
    };

    int exitcode = 0;
    int nerrors;
    nerrors = arg_parse(argc,argv,argtable);

    /* special case: '--help' takes precedence over error reporting */
    if (help->count > 0)
    {
        printf("Usage: %s", progname);
        arg_print_syntax(stdout, argtable, "\n");
        arg_print_glossary(stdout, argtable, "  %-25s %s\n");
        exitcode = 0;
        do_exit = 1;
        goto exit;
    }

    /* If the parser returned any errors then display them and exit */
    if (nerrors > 0)
    {
        /* Display the error details contained in the arg_end struct.*/
        arg_print_errors(stdout, end, progname);
        printf("Try '%s --help' for more information.\n", progname);
        exitcode = 1;
        do_exit = 1;
        goto exit;
    }
    if (timeout->count > 0) {
        args->timeout = timeout->ival[0];
    } else {
        args->timeout = 0;
    }
    if (height->count > 0) {
        args->height = height->ival[0];
    } else {
        args->height = 768;
    }
    if (width->count > 0) {
        args->width = width->ival[0];
    } else {
        args->width = 1024;
    }

    if (block_width->count > 0) {
        args->blockwidth = block_width->ival[0];
    } else {
        args->blockwidth = 32;
    }
    
    if (mode->count == 0) {
        args->mode = PROFILE_NONE;
    } else if (!strcmp(mode->sval[0], "gpu")) {
        args->mode = PROFILE_GPU;
    } else if (!strcmp(mode->sval[0], "cpu")) {
        args->mode = PROFILE_CPU;
    } else {
        printf("Unknown mode type \"%s\" \n", mode->sval[0]);
        do_exit = 1;
        exitcode = 1;
        goto exit;
    }

    args->profile = profile->count > 0;
exit:
    /* deallocate each non-null entry in argtable[] */
    arg_freetable(argtable, sizeof(argtable) / sizeof(argtable[0]));
    if (do_exit) {
        exit(exitcode);
    }
    return exitcode;
}
